#include "hip/hip_runtime.h"
//#include "../../../src/kittens.cuh"
#include "tk/src/kittens.cuh"
#include "tk/src/common/pyutils/torch_helpers.cuh"

#define NUM_WORKERS 8 // sequence parallelism
#define DIMENSION 64 // dimension of keys and values (must be a multiple of 16)

using namespace kittens; // this kernel only handles headdim=q_reg.cols for simplicity. Also n should be a multiple of 256 here.


template <typename H = bf16>
__global__ void causal_attend_kernel(
    int n,
    const H* __restrict__ __q__,
    const H* __restrict__ __k__,
    const H* __restrict__ __v__,
    const float* __restrict__ __f__,
    H* __o__
) {
    auto warpid        = kittens::warpid();
    auto block_start   = blockIdx.x*(n*DIMENSION);
    const bf16 *_q = reinterpret_cast<const bf16 *>(__q__) + block_start,
               *_k = reinterpret_cast<const bf16 *>(__k__) + block_start,
               *_v = reinterpret_cast<const bf16 *>(__v__) + block_start;
          bf16 *_o = reinterpret_cast<bf16 *>(__o__) + block_start;

    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    
    // K and V live in shared memory -- this is about all that will fit.
    st_bf_1x4<ducks::st_layout::swizzle> (&k_smem)[NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::swizzle>, NUM_WORKERS>();
    st_bf_1x4<ducks::st_layout::swizzle> (&v_smem)[NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::swizzle>, NUM_WORKERS>();

    // Initialize all of the register tiles.
    rt_bf_1x4<> q_reg, k_reg, v_reg; // v_reg need to be swapped into col_l
    rt_fl_1x1<> att_block;
    rt_bf_1x1<> att_block_mma;
    rt_fl_1x4<> o_reg;

    int qo_blocks = n / (q_reg.rows*NUM_WORKERS);

    for(auto q_blk = 0; q_blk < qo_blocks; q_blk++) {
        // each warp loads its own Q tile of 16x16
        auto q_index = q_blk*NUM_WORKERS + warpid;
        load(q_reg, _q + (q_blk*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols);

        // zero flash attention O register.
        zero(o_reg);

        // iterate over k, v for these q's that have been loaded
        for(auto kv_blk = q_blk; kv_blk >= 0; kv_blk--) {
            int kv_warp_index = kv_blk*NUM_WORKERS + warpid;
            if (kv_warp_index <= q_index) { // ensure causality
                // each warp loads its own chunk of k, v into shared memory
                load(v_smem[warpid], _v + kv_warp_index*q_reg.num_elements, q_reg.cols);
                load(k_smem[warpid], _k + kv_warp_index*q_reg.num_elements, q_reg.cols);
            }
            __syncthreads(); // we need to make sure all memory is loaded before we can begin the compute phase

            // now each warp goes through all of the subtiles, loads them, and then does the flash attention internal alg.
            for(int subtile = NUM_WORKERS-1; subtile >= 0; subtile--) {
                int kv_subtile_index = kv_blk*NUM_WORKERS + subtile;
                if (!(kv_subtile_index <= q_index)) { // ensure causality
                    continue;
                }
                load(k_reg, k_smem[subtile]); // load k from shared into registers

                zero(att_block); // zero 16x16 attention tile
                mma_ABt(att_block, q_reg, k_reg, att_block); // Q@K.T

                copy(att_block_mma, att_block); // convert to bf16 for mma_AB

                if (kv_subtile_index == q_index) {
                    make_causal(att_block_mma, att_block_mma, kittens::base_types::constants<bf16>::zero());
                }

                load(v_reg, v_smem[subtile]); // load v from shared into registers.
                rt_bf_1x4<ducks::rt_layout::col> &v_reg_col = swap_layout_inplace(v_reg); // this is a reference and the call has invalidated v_reg

                mma_AB(o_reg, att_block_mma, v_reg_col, o_reg); // mfma onto o_reg with the local attention@V matmul.
            }
            __syncthreads(); // we need to make sure all warps are done before we can start loading the next kv chunk
        }

        store(_o + (q_blk*NUM_WORKERS + warpid)*q_reg.num_elements, o_reg, q_reg.cols); // write out o. compiler has an issue with register usage if d is made constexpr q_reg.rows :/
    }
}

void
attend(torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor f, torch::Tensor o_small) {
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(f);
    CHECK_INPUT(o_small);
    
    auto batch = q.size(0);
    auto head  = q.size(1);
    auto n     = q.size(2);
    auto d     = q.size(3);
    auto dv    = v.size(3);
    bool k_same = true;
    for(auto i = 0; i < 4; i++) { 
        k_same &= q.size(i) == k.size(i);
    }
    // This is just a restriction of what we're doing now...
    TORCH_CHECK(k_same, "Q and K should be same size");
    TORCH_CHECK(q.scalar_type() == c10::ScalarType::BFloat16, "Q is a Bfloat");
    TORCH_CHECK(k.scalar_type() == c10::ScalarType::BFloat16, "K is a Bfloat");
    TORCH_CHECK(v.scalar_type() == c10::ScalarType::BFloat16, "V is a Bfloat");

    using H = __hip_bfloat16;
    using T = c10::BFloat16;
    const int workers = NUM_WORKERS;

    unsigned long mem_size = 2*workers*sizeof(st_bf_1x4<ducks::st_layout::swizzle>);

    TORCH_CHECK(n % (workers*kittens::TILE_DIM) == 0, "The number of elements should be divisible the number of workers times stored fragments");

    auto threads = workers * kittens::WARP_THREADS;
    //printf("[causal_attend] Requesting %lu bytes of memory for %d workers (%d threads)\n", mem_size, workers, threads);
    CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
             causal_attend_kernel<T>),
             hipFuncAttributeMaxDynamicSharedMemorySize, mem_size));
    
    causal_attend_kernel<T><<<batch*head,threads,mem_size>>>((int)n, q.data_ptr<T>(), k.data_ptr<T>(), v.data_ptr<T>(), f.data_ptr<float>(), o_small.data_ptr<T>());

    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

//#include "harness.impl"
    